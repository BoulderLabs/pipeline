#include "hip/hip_runtime.h"
#include "bamliquidator.h"
#include "liquidator_util.h"
#include "bamliquidator_regions.h"

#include <cmath>
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <stdexcept>
#include <string>
#include <vector>

#include <boost/lexical_cast.hpp>

#include <hdf5.h>
#include <hdf5_hl.h>

#include <tbb/blocked_range.h>
#include <tbb/enumerable_thread_specific.h>
#include <tbb/parallel_for.h>
#include <tbb/task_scheduler_init.h>

//#define time_region_parsing
#ifdef time_region_parsing 
#include <boost/timer/timer.hpp>
#endif

using namespace liquidator;

class Liquidator 
{
public:
  Liquidator(const std::string& bam_file_path):
    bam_file_path(bam_file_path),
    fp(nullptr),
    bamidx(nullptr)
  {
    init();
  }

  Liquidator(const Liquidator& other):
    bam_file_path(other.bam_file_path),
    fp(nullptr),
    bamidx(nullptr)
  {
    init();
  }

  Liquidator& operator=(const Liquidator& other) = delete;

  ~Liquidator()
  {
    bam_index_destroy(bamidx);
    samclose(fp);
  }

  double liquidate(const std::string& chromosome, int start, int stop, char strand, unsigned int extension)
  {
    std::vector<double> counts = ::liquidate(fp, bamidx, chromosome, start, stop, strand, 1, extension);
    if (counts.size() != 1)
    {
      throw std::runtime_error("liquidate failed to provide exactly one count (count is " +
        boost::lexical_cast<std::string>(counts.size()) + ")");
    }
    return counts[0];
  }

private:
  std::string bam_file_path;
  samfile_t* fp;
  bam_index_t* bamidx;

  void init()
  {
    fp = samopen(bam_file_path.c_str(),"rb",0);
    if(fp == NULL)
    {
      throw std::runtime_error("samopen() error with " + bam_file_path);
    }

    bamidx = bam_index_load(bam_file_path.c_str());
    if (bamidx == NULL)
    {
      throw std::runtime_error("bam_index_load() error with " + bam_file_path);
    }
  }
};

typedef tbb::enumerable_thread_specific<Liquidator,
                                        tbb::cache_aligned_allocator<Liquidator>,
                                        tbb::ets_key_per_instance>
        Liquidators;

void liquidate_regions(std::vector<Region>& regions, const std::string& bam_file_path,
                       size_t region_begin, size_t region_end, unsigned int extension,
                       Liquidators& liquidators)
{
  Liquidator& liquidator = liquidators.local();

  for (size_t i=region_begin; i < region_end; ++i)
  {
    try
    {
      regions[i].count = liquidator.liquidate(regions[i].chromosome,
                                              regions[i].start, 
                                              regions[i].stop, 
                                              regions[i].strand,
                                              extension);
    } catch(const std::exception& e)
    {
      Logger::error() << "Aborting because failed to parse region " << i+1 << " (" << regions[i] << ") due to error: "
                      << e.what();
      throw;
    }
  }
}

void liquidate_and_write(hid_t& file, std::vector<Region>& regions,
                         unsigned int extension, const std::string& bam_file_path)
{
  Liquidators liquidators((Liquidator(bam_file_path))); 

  tbb::parallel_for(
    tbb::blocked_range<int>(0, regions.size(), 1),
    [&](const tbb::blocked_range<int>& range)
    {
      liquidate_regions(regions, bam_file_path, range.begin(), range.end(), extension, liquidators);
    },
    tbb::auto_partitioner());

  write(file, regions);
}

int main(int argc, char* argv[])
{
  try
  {
    if (argc < 13 || argc % 2 != 1)
    {
      std::cerr << "usage: " << argv[0] << " number_of_threads region_file gff_or_bed_format extension bam_file bam_file_key hdf5_file "
                << "log_file write_warnings_to_stderr strand chr1 length1 ...\n"
        << "\ne.g. " << argv[0] << " /grail/annotations/HG19_SUM159_BRD4_-0_+0.gff gff"
        << "\n      /ifs/labs/bradner/bam/hg18/mm1s/04032013_D1L57ACXX_4.TTAGGC.hg18.bwt.sorted.bam 137 counts.hdf5 "
        << "\n      output/log.txt 1 _ chr1 247249719 chr2 242951149 chr3 199501827\n"
        << "\nstrand value of _ means use strand that is specified in region file (and use . if strand not specified in region file)."
        << "\nnumber of threads <= 0 means use a number of threads equal to the number of logical cpus."
        << "\nnote that this application is intended to be run from bamliquidator_batch.py -- see"
        << "\nhttps://github.com/BradnerLab/pipeline/wiki for more information"
        << std::endl;
      return 1;
    }

    const int number_of_threads = boost::lexical_cast<int>(argv[1]);
    const std::string region_file_path = argv[2];
    const std::string region_format = argv[3];
    const unsigned int extension = boost::lexical_cast<unsigned int>(argv[4]);
    const std::string bam_file_path = argv[5];
    const unsigned int bam_file_key = boost::lexical_cast<unsigned int>(argv[6]);
    const std::string hdf5_file_path = argv[7];
    const std::string log_file_path = argv[8];
    const bool write_warnings_to_stderr = boost::lexical_cast<bool>(argv[9]);
    const char strand = boost::lexical_cast<char>(argv[10]);
    const std::vector<std::pair<std::string, size_t>> chromosome_lengths = extract_chromosome_lengths(argc, argv, 11);

    tbb::task_scheduler_init init( number_of_threads <= 0 
                                 ? tbb::task_scheduler_init::automatic
                                 : number_of_threads); 

    Logger::configure(log_file_path, write_warnings_to_stderr);

    hid_t h5file = H5Fopen(hdf5_file_path.c_str(), H5F_ACC_RDWR, H5P_DEFAULT);
    if (h5file < 0)
    {
      Logger::error() << "Failed to open H5 file " << hdf5_file_path;
      return 3;
    }

    #ifdef time_region_parsing 
    boost::timer::cpu_timer timer; 
    #endif

    std::map<std::string, size_t> chromosome_to_length;
    for (auto& chr_length : chromosome_lengths)
    {
      chromosome_to_length[chr_length.first] = chr_length.second;
    }

    std::vector<Region> regions = parse_regions(region_file_path,
                                                region_format,
                                                bam_file_key,
                                                chromosome_to_length,
                                                strand);
    #ifdef time_region_parsing 
    timer.stop();
    std::cout << "parsing regions took" << timer.format() << std::endl;
    #endif
    if (regions.size() == 0)
    {
      Logger::warn() << "No valid regions detected in " << region_file_path;
      return 0;
    }

    liquidate_and_write(h5file, regions, extension, bam_file_path);
   
    H5Fclose(h5file);

    return 0;
  }
  catch(const std::exception& e)
  {
    Logger::error() << "Unhandled exception: " << e.what();

    return 4; 
  }
}

/* The MIT License (MIT) 

   Copyright (c) 2013 John DiMatteo (jdimatteo@gmail.com)

   Permission is hereby granted, free of charge, to any person obtaining a copy
   of this software and associated documentation files (the "Software"), to deal
   in the Software without restriction, including without limitation the rights
   to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
   copies of the Software, and to permit persons to whom the Software is
   furnished to do so, subject to the following conditions:

   The above copyright notice and this permission notice shall be included in
   all copies or substantial portions of the Software.

   THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
   IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
   FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
   AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
   LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
   OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
   THE SOFTWARE. 
 */
